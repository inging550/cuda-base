#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__global__ void gpu_share_memory(float *d_a)
{
	int index = threadIdx.x;
	float average, sum = 0.0f;
	__shared__ float sh_arr[10];
	sh_arr[index] = d_a[index];
	__syncthreads();  // 确保赋值完成再进行计算
	for (int i = 0; i <= index; i++)
	{
		sum += sh_arr[i];
	}
	average = sum / (index + 1.0f);
	d_a[index] = average;
}


int main()
{
	float h_a[10];
	float *d_a;
	for (int i = 0; i < 10; i++)
	{
		h_a[i] = i;
	}
	hipMalloc((void**)&d_a, sizeof(float) * 10);
	hipMemcpy((void*)d_a, (void*)h_a, sizeof(float) * 10, hipMemcpyHostToDevice);
	// 开始计算
	hipEvent_t e_start, e_end; // 建立事件对象
	hipEventCreate(&e_start); // 建立事件
	hipEventCreate(&e_end);
	hipEventRecord(e_start, 0);  // 记录时间戳

	gpu_share_memory << <1, 10 >> > (d_a);

	hipDeviceSynchronize();  // 等待核函数执行完毕
	hipEventRecord(e_end, 0);  // 等待记录命令执行完毕
	hipEventSynchronize(e_end);  // 记录时间戳
	float Time;
	hipEventElapsedTime(&Time, e_start, e_end);  // 计算时间
	printf("Time:%f ms\n", Time);
	hipMemcpy((void*)h_a, (void*)d_a, sizeof(float)*10, hipMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++)
	{
		printf("%d--->%f\n", i, h_a[i]);
	}
	return 0;
}
